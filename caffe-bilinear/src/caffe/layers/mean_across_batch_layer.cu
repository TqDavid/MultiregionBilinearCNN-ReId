#include "hip/hip_runtime.h"
#include <cfloat>
#include <vector>

#include "caffe/layers/mean_across_batch_layer.hpp"
#include "caffe/util/math_functions.hpp"

namespace caffe {



template <typename Dtype>
void MeanAcrossBatchLayer<Dtype>::Forward_gpu(
    const vector<Blob<Dtype>*>& bottom, const vector<Blob<Dtype>*>& top) {

  int chunk_size = this->layer_param().mean_across_batch_param().chunk_size();
  int count = bottom[0]->channels() * bottom[0]->height() * bottom[0]->width();
  
  for (int i = 0; i < (bottom[0]->num() + chunk_size)/chunk_size; i++){
      int s = i * chunk_size;
      int e = std::min((i + 1) * chunk_size, bottom[0]->num());
      
      if (s >= e)
	  break;

      caffe_gpu_set(count, Dtype(0.), mean_in_chunk_.mutable_gpu_data());
      for (int j = s; j < e; j++){
          caffe_gpu_axpy(count, Dtype(1./(e-s)), bottom[0]->gpu_data() + count * j, mean_in_chunk_.mutable_gpu_data());

      }
      for (int j = s; j < e; j++){
          caffe_copy(count, mean_in_chunk_.gpu_data(), top[0]->mutable_gpu_data() + count * j);
      }

  }


}

template <typename Dtype>
void MeanAcrossBatchLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
    const vector<bool>& propagate_down, const vector<Blob<Dtype>*>& bottom) {

  int chunk_size = this->layer_param().mean_across_batch_param().chunk_size();
  int count = bottom[0]->channels() * bottom[0]->height() * bottom[0]->width(); 
  
  for (int i = 0; i < (top[0]->num() + chunk_size)/chunk_size; i++){
      int s = i * chunk_size;
      int e = std::min((i + 1) * chunk_size, bottom[0]->num());
      
      if (s >= e)
	  break;

      caffe_gpu_set(count, Dtype(0.), mean_in_chunk_.mutable_gpu_diff());
      for (int j = s; j < e; j++){
          caffe_gpu_axpy(count, Dtype(1./(e-s)), top[0]->gpu_diff() + count * j, mean_in_chunk_.mutable_gpu_diff());
      }
      for (int j = s; j < e; j++){
          caffe_copy(count, mean_in_chunk_.gpu_diff(), bottom[0]->mutable_gpu_diff() + count * j);
      }
  }
}

INSTANTIATE_LAYER_GPU_FUNCS(MeanAcrossBatchLayer);


}  // namespace caffe
